#include "hip/hip_runtime.h"
#include "matrix_mul.h"
#include <sys/time.h>
#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>
#include <math.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/sequence.h>
#include <thrust/fill.h>
#include <thrust/transform.h>
#define MAX_THREADS_PER_BLOCK 32
namespace kernel{
template<typename T>
__global__ void matrixMultiplication(const T *a, const T *b, T *c,
        bool transpose_a, bool transpose_b, int a_rows, int a_cols, int b_rows, int b_cols) {

    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
    printf("ROW: %d blockIdx.y: %d blockDim.y %d threadIdx.y: %d\n", ROW, blockIdx.y, blockDim.y, threadIdx.y);
    int COL = blockIdx.x*blockDim.x+threadIdx.x;
    printf("COL: %d blockIdx.x: %d blockDim.x %d threadIdx.x: %d\n", ROW, blockIdx.x, blockDim.x, threadIdx.y);

    int c_rows = transpose_a ? a_cols : a_rows;
    int shared = transpose_a ? a_rows : a_cols;
    int c_cols = transpose_b ? b_rows : b_cols;

    if (ROW < c_rows && COL < c_cols) {
        for (int k = 0; k < shared; k++) {

            int a_idx = transpose_a ? k * a_cols + ROW : ROW * a_cols + k;
            int b_idx = transpose_b ? COL * b_cols + k : k * b_cols + COL;                
            c[ROW*c_cols + COL] += a[a_idx] * b[b_idx];
        }
    }
}
}

namespace gpu {
template<typename T, typename I>
void matrixMultiplication(
        const thrust::device_vector<T, I> *a, const thrust::device_vector<T, I> *b, thrust::device_vector<T, I> *c,
        bool transpose_a, bool transpose_b,
        size_t a_rows, size_t a_cols, size_t b_rows, size_t b_cols) {

    size_t rows = transpose_a ? a_cols : a_rows;

    size_t shared = transpose_a ? a_rows : a_cols;
    assert(shared == (transpose_b ? b_cols : b_rows));

    size_t cols = transpose_b ? b_rows : b_cols;

    printf("matmul: %dx%dx%d\n", rows, shared, cols);

    dim3 threadsPerBlock(cols, rows);
    dim3 blocksPerGrid(1, 1);

    if (cols > MAX_THREADS_PER_BLOCK) {
        threadsPerBlock.x = MAX_THREADS_PER_BLOCK;
        blocksPerGrid.x = ceil(double(cols)/double(threadsPerBlock.x));
    }
    
    if (rows > MAX_THREADS_PER_BLOCK) {
        threadsPerBlock.y = MAX_THREADS_PER_BLOCK;
        blocksPerGrid.y = ceil(double(rows)/double(threadsPerBlock.y));
    }

    kernel::matrixMultiplication<<<blocksPerGrid,threadsPerBlock>>>(
        thrust::raw_pointer_cast(&a->begin()[0]),
        thrust::raw_pointer_cast(&b->begin()[0]),
        thrust::raw_pointer_cast(&c->begin()[0]),
        transpose_a, transpose_b, a_rows, a_cols, b_rows, b_cols
    );

    hipDeviceSynchronize();
}
}


template<typename T>
void matrixMultiplication(
    std::vector<T> a, std::vector<T> b, std::vector<T> c,
    bool transpose_a, bool transpose_b,
    size_t a_rows, size_t a_cols, size_t b_rows, size_t b_cols){
        thrust::device_vector<T> gpu_a(a.begin(), a.end());
        thrust::device_vector<T> gpu_b(b.begin(), b.end());
        thrust::device_vector<T> gpu_c(c.begin(), c.end());

        gpu::matrixMultiplication(
            &gpu_a, &gpu_b, &gpu_c,
            transpose_a, transpose_b, a_rows, a_cols, b_rows, b_cols
        );
    }

template void matrixMultiplicationWrapper(std::vector<uint64_t> a, std::vector<uint64_t> b, std::vector<uint64_t> c, bool transpose_a, 
    bool transpose_b, size_t a_rows, size_t a_cols, size_t b_rows, size_t b_cols);

template void matrixMultiplicationWrapper(std::vector<uint32_t> a, std::vector<uint32_t> b, std::vector<uint32_t> c, bool transpose_a, 
    bool transpose_b, size_t a_rows, size_t a_cols, size_t b_rows, size_t b_cols);

template void matrixMultiplicationWrapper(std::vector<uint16_t> a, std::vector<uint16_t> b, std::vector<uint16_t> c, bool transpose_a, 
    bool transpose_b, size_t a_rows, size_t a_cols, size_t b_rows, size_t b_cols);

template void matrixMultiplicationWrapper(std::vector<uint8_t> a, std::vector<uint8_t> b, std::vector<uint8_t> c, bool transpose_a, 
    bool transpose_b, size_t a_rows, size_t a_cols, size_t b_rows, size_t b_cols);

template void matrixMultiplicationWrapper(std::vector<int> a, std::vector<int> b, std::vector<int> c, bool transpose_a, 
    bool transpose_b, size_t a_rows, size_t a_cols, size_t b_rows, size_t b_cols);

template void matrixMultiplicationWrapper(std::vector<float> a, std::vector<float> b, std::vector<float> c, bool transpose_a, 
    bool transpose_b, size_t a_rows, size_t a_cols, size_t b_rows, size_t b_cols);


