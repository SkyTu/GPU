#include "hip/hip_runtime.h"
#include "foo.cuh"

__global__ void helloworld(){
    printf("Hello World from the gpu");
}

void helloworldwrapper(){
    helloworld<<<1,1>>>();
    hipDeviceSynchronize();
}